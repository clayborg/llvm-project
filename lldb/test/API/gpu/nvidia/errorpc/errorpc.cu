
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

// Super simple assert test. Hits an assertion on device side

static void VerifyCudaError(hipError_t err, const char *file, int line,
                            hipError_t expected_err) {
  if (err != expected_err) {
    fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file,
            line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_CUDA_ERROR(err)                                                 \
  (VerifyCudaError(err, __FILE__, __LINE__, hipSuccess))

#define N 256

__global__ void fault() {
  int fault_here = *(volatile int *)0x03;

  if (fault_here)
    printf("Fault here\n");
}

int main(void) {
  hipDeviceReset();
  fault<<<20, N>>>(); // breakpoint1
  HANDLE_CUDA_ERROR(hipDeviceSynchronize());
  HANDLE_CUDA_ERROR(hipGetLastError());

  return 0;
}
